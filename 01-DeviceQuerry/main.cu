
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int numDevices;
    hipGetDeviceCount(&numDevices);
    printf("Num devices = %d\n", numDevices);
    for (int i = 0; i < numDevices; i++) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("Device %i is %s and has %d multiprocessors\n", i, props.name, props.multiProcessorCount);
    }
}